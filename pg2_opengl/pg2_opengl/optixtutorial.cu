#include "hip/hip_runtime.h"
#include "optixtutorial.h"

struct TriangleAttributes
{
	optix::float3 normal;
	optix::float2 texcoord;
	optix::float3 intersectionPoint;
	optix::float3 vectorToLight;
};

rtBuffer<optix::float3, 1> normal_buffer;
rtBuffer<optix::uchar4, 2> output_buffer;

rtDeclareVariable( rtObject, top_object, , );
rtDeclareVariable( uint2, launch_dim, rtLaunchDim, );
rtDeclareVariable( uint2, launch_index, rtLaunchIndex, );
rtDeclareVariable( PerRayData_radiance, shadow_ray_data, rtPayload, );
rtDeclareVariable( float2, barycentrics, attribute rtTriangleBarycentrics, );
rtDeclareVariable(TriangleAttributes, attribs, attribute attributes, "Triangle attributes");
rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );
rtDeclareVariable(optix::float3, view_from, , );


RT_PROGRAM void attribute_program(void)
{
	const optix::float3 lightPossition = optix::make_float3(100, 100, 200);
	const optix::float2 barycentrics = rtGetTriangleBarycentrics();
	const unsigned int index = rtGetPrimitiveIndex();
	const optix::float3 n0 = normal_buffer[index * 3 + 0];
	const optix::float3 n1 = normal_buffer[index * 3 + 1];
	const optix::float3 n2 = normal_buffer[index * 3 + 2];

	attribs.normal = optix::normalize(n1 * barycentrics.x + n2 * barycentrics.y + n0 * (1.0f - barycentrics.x - barycentrics.y));

	if (optix::dot(ray.direction, attribs.normal) > 0) {
		attribs.normal *= -1;
	}

	attribs.intersectionPoint = optix::make_float3(ray.origin.x + ray.tmax * ray.direction.x,
		ray.origin.y + ray.tmax * ray.direction.y,
		ray.origin.z + ray.tmax * ray.direction.z);

	attribs.vectorToLight = lightPossition - attribs.intersectionPoint;
}

RT_PROGRAM void primary_ray( void )
{

	optix::Ray ray(view_from, attribs.vectorToLight, 0, 0.01f);

	PerRayData_radiance prd;
	rtTrace(top_object, ray, prd);

	//prd.visible should be set
	// access to buffers within OptiX programs uses a simple array syntax	
	output_buffer[launch_index] = optix::make_uchar4(prd.attenuation.x*255.0f, prd.attenuation.y*255.0f, prd.attenuation.z*255.0f, 255);
}



RT_PROGRAM void any_hit(void)
{
	shadow_ray_data.attenuation  = optix::make_float3(0.5f,0.5f,0.5f);
	rtTerminateRay();
}



/* may access variables declared with the rtPayload semantic in the same way as closest-hit and any-hit programs */
RT_PROGRAM void miss_program( void )
{
	shadow_ray_data.visible = 0.0f;
}

RT_PROGRAM void exception( void )
{
	const unsigned int code = rtGetExceptionCode();
	rtPrintf( "Exception 0x%X at (%d, %d)\n", code, launch_index.x, launch_index.y );
	rtPrintExceptionDetails();
	output_buffer[launch_index] = uchar4{ 255, 0, 255, 0 };
}


